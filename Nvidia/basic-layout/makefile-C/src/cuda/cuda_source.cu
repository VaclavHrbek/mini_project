#include "hip/hip_runtime.h"
#include "cuda/cuda_header.h"

__global__
void device_function(int* var){
	(*var)++;
}

// Wrapper around cuda kernel
extern "C"
void cuda_function(int var){
	int* i;
	int* d_var;
	i = (int*)malloc(sizeof(int));
	hipMalloc(&d_var, sizeof(int));

	*i = var;

	hipMemcpy(d_var, i, sizeof(int), hipMemcpyHostToDevice);
	device_function<<<1, 1>>>(d_var);
	hipMemcpy(i, d_var, sizeof(int), hipMemcpyDeviceToHost);

	printf("From .cu file. Kernel returned: %d\n", *i);

	hipFree(d_var);
	free(i);
}

