#include "hip/hip_runtime.h"
#include "cuda/cuda_header.cuh"

__global__
void device_function(int* var){
	(*var)++;
}

// Wrapper around cuda kernel
void cuda_function(int var){
	int* i;
	int* d_var;
	i = (int*)malloc(sizeof(int));
	hipMalloc(&d_var, sizeof(int));

	*i = var;

	hipMemcpy(d_var, i, sizeof(int), hipMemcpyHostToDevice);
	device_function<<<2, 3>>>(d_var);
	hipMemcpy(i, d_var, sizeof(int), hipMemcpyDeviceToHost);

	printf("From .cu file. Kernel returned: %d\n", *i);

	hipFree(d_var);
	free(i);
}


void cuda_function_call_c_function(){
	c_function(3);
}

