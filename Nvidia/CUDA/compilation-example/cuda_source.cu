#include "hip/hip_runtime.h"
#include "cuda_header.h"

__global__
void cuda_function(int* var){
	(*var)++;
}

extern "C"
void function(int var){
	int* i;
	int* d_var;
	i = (int*)malloc(sizeof(int));
	hipMalloc(&d_var, sizeof(int));

	*i = var;

	hipMemcpy(d_var, i, sizeof(int), hipMemcpyHostToDevice);
	cuda_function<<<1, 1>>>(d_var);
	hipMemcpy(i, d_var, sizeof(int), hipMemcpyDeviceToHost);

	printf("Number from the kernel: %d\n", *i);

	hipFree(d_var);
	free(i);
}

