#include <vector>
#include <iostream>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <nvToolsExt.h>

class Data {
	int _val;
	public:
	Data(int a) : _val(a) {}
	__device__
	void plus_one() {
		_val += 1;
	};
	int val(){
		return _val;
	}
};

struct printf_functor
{
  __host__ __device__
  void operator()(int x)
  {
    // note that using printf in a __device__ function requires
    // code compiled for a GPU with compute capability 2.0 or
    // higher (nvcc --arch=sm_20)
    printf("%d\n", x);
  }
};

int main(){
	thrust::host_vector<Data> vec;
	nvtxRangePush("Seq init data");
	for(size_t i = 0; i != 10000; i++){
		vec.push_back(Data(i));	
	}
	nvtxRangePop();

	nvtxRangePush("Copy data to device + malloc on device");
	thrust::device_vector<Data> d_vec = vec;
	nvtxRangePop();

	nvtxRangePush("Call data on GPU");
	thrust::for_each(thrust::device, d_vec.begin(), d_vec.end(), []__device__ (Data &x){x.plus_one();});
	hipDeviceSynchronize();
	nvtxRangePop();
	//thrust::copy(d_vec.begin(), d_vec.end(), vec.begin());
	nvtxRangePush("Copy data to host");
	vec = d_vec;
	nvtxRangePop();
//	for(auto i : vec){
//		std::cout << i.val() << ' ';
//	}

	return 0;
}
