#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

__global__
void foo_kernel(int* a, int* b, size_t* size){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < *size){
		//a[id] = b[id];
	}
}

int main(void){
	size_t size = 10;
	size_t* ptr_size;
	int* a = (int*)(malloc(sizeof(int)*size));
	int* b = (int*)(malloc(sizeof(int)*size));
	int* d_a;
	int* d_b;
	hipMalloc(&d_a, (sizeof(int)*size));
	hipMalloc(&d_b, (sizeof(int)*size));
	hipMalloc(&ptr_size, (sizeof(size_t)));
	for(size_t i = 0; i != size; ++i){
		b[i] = i;
	}
	hipMemcpy(d_b, b, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(ptr_size, &size, sizeof(size_t), hipMemcpyHostToDevice);

	// create generation type
	hiprandGenerator_t gen;
	// create generator
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	// seed the generator
	hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
	// generate numbers (Generate 32-bit pseudo or quasirandom numbers)
	// generate numbers on device
	hiprandGenerate(gen, (uint*)d_a, size);

	foo_kernel<<<1,1024>>>(d_a, d_b, ptr_size);
	hipMemcpy(a, d_a, sizeof(int)*size, hipMemcpyDeviceToHost);
	
	// error checking
	int err;
	for(size_t i = 0; i != size; ++i){
		err += a[i] - i;
		printf("%i\n", a[i]);
	}
	if(err != 0){
		printf("The computation is wrong\n");
	}
	else{
		printf("The computation is success\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	free(a);
	free(b);
	hipFree(ptr_size);
}

